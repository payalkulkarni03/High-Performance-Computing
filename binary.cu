#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cstring>
#include <ctime>
#include <math.h>
#define N 1024*40000
__device__ int binarySearch1(float *, int , int , int );
int binarySearch(float *, int , int , int );



__global__ void binary (float *Array, float *A2,float  key ,int size)   //Kernel Code For Reduction
 {
   	//holds intermediates in shared memory rr
   	int result;

    	int i = blockIdx.x * blockDim.x + threadIdx.x;

      int split=4;

     
     
        if(key>Array[(size/split)*i]&&key<Array[(size/split)*(i+1)])
        {
        A2[0]=(size/split)*i;  //low
        A2[1]=(size/split)*(i+1); //high
        result=binarySearch1(Array,A2[0],A2[1],key);
        A2[2]=result; //high - low
       
        
      
    }

}

__device__ int binarySearch1(float *arr, int l, int r, int x)
{
   if (r >= l)
   {
        int mid = l + (r - l)/2;

        // If the element is present at the middle
        // itself
        if (arr[mid] == x)
            return mid;

        // If element is smaller than mid, then
        // it can only be present in left subarray
        if (arr[mid] > x)
            return binarySearch1(arr, l, mid-1, x);

        // Else the element can only be present
        // in right subarray
        return binarySearch1(arr, mid+1, r, x);
   }

   // We reach here when element is not
   // present in array
   return -1;
}
int binarySearch(float *arr, int l, int r, int x)
{
   if (r >= l)
   {
        int mid = l + (r - l)/2;

        // If the element is present at the middle
        // itself
        if (arr[mid] == x)
            return mid;

        // If element is smaller than mid, then
        // it can only be present in left subarray
        if (arr[mid] > x)
            return binarySearch(arr, l, mid-1, x);

        // Else the element can only be present
        // in right subarray
        return binarySearch(arr, mid+1, r, x);
   }

   // We reach here when element is not
   // present in array
   return -1;
}


int main()
{

	size_t size = N * sizeof(float);
	clock_t start,stop; //to measure time of excecution
  printf("\nName of the Model= Parllel Binary Search\n");
//Thread allocation
  int threadsPerBlock;
  if (N<=1024)
  		threadsPerBlock=1;
  else
		 threadsPerBlock=N/1024;
  int blocksPerGrid =(N + threadsPerBlock - 1) / threadsPerBlock;
  printf("\nblocksPerGrid=%d\n",blocksPerGrid);
// Memory Allocation
  float* device_Array; //input array
  float* device_output;
  float result;
  float *host_out = (float *) malloc(3 * sizeof(float));
  float* host_Array = (float*)malloc(size);				// Allocate input vectors h_A and h_B in host memory
	float host_key  ;
  host_key=(float)50;

  hipMalloc(&device_Array, size);
  hipMalloc(&device_output,3*sizeof(float));						// Allocate vector in device memory

  FILE *f;
	f=fopen("Binary.txt","a"); //to store the result in to file


	for(int i = 0; i < N; i++) {					// Initialize input vectors
        	host_Array[i] = i;//rand()%100;
        //  printf("%f\n",host_Array[i] );
    	}
   
 /* for(int i = 0; i < N; i++) {
     //  printf("%d\t",i);
       printf("%f\n",host_Array[i] );
       fprintf(f,"\t\t%d\t",i );
       fprintf(f,"%f\n",host_Array[i] );
    	}*/
//Actual Logic
 
  hipMemcpy(device_Array, host_Array, size, hipMemcpyHostToDevice); //copy data to GPU
   start = std::clock();
  binary<<<1,4>>>(device_Array,device_output,host_key,N); // Invoke kernel
  stop = std::clock();
  hipMemcpy(host_out,device_output, 3*sizeof(float), hipMemcpyDeviceToHost);//copy to CPU
  
	
  long int GPU_time=stop - start;
  printf("Start of Partition   \t%f\n",host_out[0] );
  printf("End of Partition     \t%f\n",host_out[1]);
 
  printf("_______________________________________________________________________	\n\n"); //print to console
  printf("Result By GPU= %f ",host_out[2]);
  printf("\n\nExecution GPU_time of parllel Implementation= %ld (ms)\n", GPU_time );
  printf("_______________________________________________________________________	\n");

  fprintf(f,"_______________________________________________________________________	\n\n"); //print to file
  fprintf(f,"\t\tResult By GPU= %f \n\n ",host_out[2]);
  fprintf(f,"\n\n\t\tExecution GPU_time of parllel Implementation= %ld (ms)\n\n", GPU_time );
  fprintf(f,"_______________________________________________________________________\n	");

  start = std::clock();
  result= binarySearch( host_Array,0,N,host_key); // Calculation by cpu
  stop = std::clock();
  long int CPU_time=stop - start;
  printf("\nCPU Result= %f ",result);
  printf("\n\nExecution Time of Sequential Implementation= %ld (ms)\n",CPU_time );
  printf("_______________________________________________________________________	");

  fprintf(f,"\n\t\tCPU Result= %f ",result);                                                //cpu result print in file
  fprintf(f,"\n\n\t\tExecution Time of Sequential Implementation= %ld (ms)\n",CPU_time );
  fprintf(f,"_______________________________________________________________________	");

  float eff=float(CPU_time)/float(GPU_time);
  printf("\n\nSpeedup=CPU_TIME / GPU_TIME  =  %f\n",eff);
	printf("_______________________________________________________________________	");

  fprintf(f,"\n\nSpeedup=CPU_TIME / GPU_TIME  =  %f\n",eff);
  fprintf(f,"_______________________________________________________________________	");

  // Free device memory
	hipFree(device_Array);
	hipFree(device_output);

  // Free host memory
  free(host_Array);
  free(host_out);
}
