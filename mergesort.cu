
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define N 10
// helper for main()
long readList(long**);

// data[], size, threads, blocks, 
void mergesort(long*, long, dim3, dim3);
// A[]. B[], size, width, slices, nThreads
__global__ void gpu_mergesort(long*, long*, long, long, long, dim3*, dim3*);
__device__ void gpu_bottomUpMerge(long*, long*, long, long, long);
int tm();



#define min(a, b) (a < b ? a : b)

void printHelp(char* program) {

    
}


int main(int argc, char** argv) {

    dim3 threadsPerBlock;
    dim3 blocksPerGrid;

    threadsPerBlock.x = 32;
    

    blocksPerGrid.x = 8;
    


    long* data = (long*)malloc(N*sizeof(long));;

   for(int i = 0; i < N; i++) {
       data[i] = rand()%100;
}

    
         printf("sorting %d numbers\n",N);

    // merge-sort the data
    mergesort(data, N, threadsPerBlock, blocksPerGrid);

   

    
    for (int i = 0; i < N; i++) {
         printf("%ld\n", data[i] );
    } 

   
      
   
}

void mergesort(long* data, long size, dim3 threadsPerBlock, dim3 blocksPerGrid) {

    
    // Allocate two arrays on the GPU
   
    
    long* D_data;
    long* D_swp;
    dim3* D_threads;
    dim3* D_blocks;
    
    
    hipMalloc((void**) &D_data, size * sizeof(long));
   hipMalloc((void**) &D_swp, size * sizeof(long));
   
       

    // Copy from our input list into the first array
    hipMemcpy(D_data, data, size * sizeof(long), hipMemcpyHostToDevice);
 
      
 
  
    // Copy the thread / block info to the GPU as well
   
    hipMalloc((void**) &D_threads, sizeof(dim3));
    hipMalloc((void**) &D_blocks, sizeof(dim3));

   
      
   hipMemcpy(D_threads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice);
    hipMemcpy(D_blocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice);

   
       

    long* A = D_data;
    long* B = D_swp;

    long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                    blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

    //
    // Slice up the list and give pieces of it to each thread, letting the pieces grow
    // bigger and bigger until the whole list is sorted
    //
    for (int width = 2; width < (size << 1); width <<= 1) {
        long slices = size / ((nThreads) * width) + 1;

        
         
        gpu_mergesort<<<blocksPerGrid, threadsPerBlock>>>(A, B, size, width, slices, D_threads, D_blocks);

        // Switch the input / output arrays instead of copying them around
        A = A == D_data ? D_swp : D_data;
        B = B == D_data ? D_swp : D_data;
    }

    //
    // Get the list back from the GPU
    //
    
    hipMemcpy(data, A, size * sizeof(long), hipMemcpyDeviceToHost);
   
    
    
    
    // Free the GPU memory
  hipFree(A);
    hipFree(B);
   
      
}

__device__ void gpu_Merge(long* source, long* dest, long start, long middle, long end) {
    long i = start;
    long j = middle;
    for (long k = start; k < end; k++) {
        if (i < middle && (j >= end || source[i] < source[j])) {
            dest[k] = source[i];
            i++;
        } else {
            dest[k] = source[j];
            j++;
        }
    }
}


//
// Perform a full mergesort on our section of the data.
//
__global__ void gpu_mergesort(long* source, long* dest, long size, long width, long slices, dim3* threads, dim3* blocks) {
   
	unsigned int idx=(blockIdx.x*blockDim.x)+threadIdx.x;
    	long start = width*idx*slices,
         middle, 
         end;

    for (long slice = 0; slice < slices; slice++) {
        if (start >= size)
            break;

        middle = min(start + (width >> 1), size);
        end = min(start + width, size);
        gpu_Merge(source, dest, start, middle, end);
        start += width;
    }
}

//
// Finally, sort something
// gets called by gpu_mergesort() for each slice
//

// read data into a minimal linked list

     
    





