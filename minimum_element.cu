
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
//	#include<omp.h>
#define SIZE 1024
__global__ void min(int * A, int * C)
{
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	A[2*i]<A[2*i+1]?C[i]=A[2*i]:C[i]=A[2*i+1];
			
}
int main()
{
	int A[SIZE];
	int *devA,*devC;
	//double start,end;
	for(int j=0;j<SIZE;j++)
	{
		A[j]=SIZE-j;
	}
	hipMalloc((void **)&devA,SIZE*sizeof(int));
	hipMalloc((void **)&devC,SIZE*sizeof(int));
	//start=omp_get_wtime();
	//printf("\nStart time:%f",start);
	for(int j=1;j<log2((double)SIZE);j++)
	{
		hipMemcpy(devA,A,SIZE*sizeof(int),hipMemcpyHostToDevice);
		min<<<1,SIZE/pow(2,j)>>>(devA,devC);
		hipMemcpy(&A,devC,SIZE*sizeof(int),hipMemcpyDeviceToHost);
	}
	//end=omp_get_wtime();
	//printf("\nEnd time:%f",end);
	//printf("\nTotal time:%f\n",end-start);
	A[0]<A[1]?printf("\nMin is:%d\n",A[0]):printf("\nMin is:%d\n",A[1]);
	hipFree(devA);
	hipFree(devC);
	return 0;
}
