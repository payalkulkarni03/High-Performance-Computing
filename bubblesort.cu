
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<omp.h>
#define SIZE 1024
__global__ void sort(int * A, int j)
{
        int temp=0;
        int i=blockIdx.x*blockDim.x+threadIdx.x;
        if(j%2==0)
        {
                if(A[2*i]>A[2*i+1])
                {
                        temp=A[2*i];
                        A[2*i]=A[2*i+1];
                        A[2*i+1]=temp;
                }
        }
        else
        {
                if(A[2*i+1]>A[2*i+2])
                {
                        temp=A[2*i+1];
                        A[2*i+1]=A[2*i+2];
                        A[2*i+2]=temp;
                }
        }

}
int main()
{
        int A[SIZE];
        int *devA;
       // double start,end;
        for(int j=0;j<SIZE;j++) //initialize array
        {
                A[j]=SIZE-j;
        }
 
        hipMalloc((void **)&devA,SIZE*sizeof(int)); //allocate memory to gpu devices
        
        //calculate start time 
       // start=omp_get_wtime();
        //printf("\nStart time:%f",start);

	hipMemcpy(devA,A,SIZE*sizeof(int),hipMemcpyHostToDevice);
        for(int j=0;j<(SIZE);j++)
        {

                if(j%2==0)
                {
                        sort<<<1,SIZE/2>>>(devA,j);
                }
                else
                        sort<<<1,((SIZE/2)-1)>>>(devA,j);

        }
        hipMemcpy(&A,devA,SIZE*sizeof(int),hipMemcpyDeviceToHost);
     
        //calculate end time
	//end=omp_get_wtime();
        //printf("\nEnd time:%f",end);
        //printf("\nTotal time:%f\n",end-start);

        printf("Sorted array is:\n");
        for(int i=0;i<SIZE;i++)
        {
                printf("\t%d",A[i]);
        }
        hipFree(devA);

        return 0;
}


