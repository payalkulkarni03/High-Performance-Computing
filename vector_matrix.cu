#include <stdio.h>
#include<hip/hip_runtime.h>
#include <stdlib.h>
#include<time.h>

#define SIZE 45

__global__ void matrixvectmult(int *mat,int *vect,int *res)
{
	int tid=blockIdx.x*blockDim.x;
	int mult=0;
	for(int i=0;i<SIZE;i++)
	{
		mult=mult+(mat[tid+i]*vect[i]);
	}
	res[blockIdx.x]=mult;
}

int main(void)
{
	int i,j;
	srand(time(NULL));
	int a[SIZE][SIZE],b[SIZE],c[SIZE];

	int *dev_a,*dev_b,*dev_c;

	hipMalloc((void **)&dev_a, SIZE*SIZE*sizeof(int));
	hipMalloc((void **)&dev_b, SIZE*sizeof(int));
	hipMalloc((void **)&dev_c, SIZE*sizeof(int));

	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			a[i][j] = rand()%20+1;
		}
	}

	printf("\nThe matrix is:\n");
	for(i=0;i<SIZE;i++)
	{
		for(j=0;j<SIZE;j++)
		{
			printf("%d\t",a[i][j]);
		}
		printf("\n");
	}

	for(i=0;i<SIZE;i++)
	{
		b[i] = rand()%20+1;
	}

	printf("\nThe vector is:\n");
	for(i=0;i<SIZE;i++)
	{
		printf("%d  ",b[i]);
	}

	hipMemcpy(dev_a,a,sizeof(a),hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,sizeof(b),hipMemcpyHostToDevice);
	matrixvectmult<<<SIZE,SIZE>>>(dev_a,dev_b,dev_c);
	hipMemcpy(&c,dev_c,sizeof(c),hipMemcpyDeviceToHost);

	printf("\nThe result is:\n");
	for(int i=0;i<SIZE;i++)
	{
		printf("%d ",c[i]);
	}


	return 0;
}
